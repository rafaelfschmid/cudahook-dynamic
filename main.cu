/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>
#include <thread>
#include <chrono>
#include "Scheduler.h"

void exec(const char* s){
	system(s);
}
//hipStream_t streams[NUM_STREAMS];

int main(int argc, char **argv) {

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	std::string line = "";
	std::getline (std::cin, line);
	exec(line.data());
	//s.add(line);
	//std::thread t1(exec,line.data());

	std::string line2 = "";
	std::getline (std::cin, line2);
	//exec(line.data());
	exec(line2.data());
	//std::thread t2(exec,line2.data());

	//rodinia::main(argc, argv);

	//Scheduler s;
	//int *num = Scheduler::num;
	//s.init(0);
	//try{}
	//catch(...){printf("Exception\n");}
	/*
	std::string line = "";
	while(line != " ") {
		std::getline (std::cin, line);
		//std::cout << line << "\n";
		std::string str = argv[i];//"./hotspot 1024 2 2 ../../data/hotspot/temp_1024 ../../data/hotspot/power_1024 output.out";
		s.programCall(line);
	//	s.schedule();
	}*/
	//s.execute();

	return 0;
}
